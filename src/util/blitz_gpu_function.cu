#include "util/blitz_gpu_function.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>

namespace blitz {

boost::scoped_ptr<hipblasHandle_t> CuBlasHandle::instance_(0);
boost::once_flag CuBlasHandle::flag_ = BOOST_ONCE_INIT;

template<>
void BlitzGPUGemm(const bool transa, const bool transb,
  const int M, const int N, const int K,
  float* A, float* B, float* C, float alpha, float beta) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int ldb = transb ? K : N;
  hipblasSgemm(CuBlasHandle::GetInstance(), TransB, TransA, N, M, K,
    &alpha, B, ldb, A, lda, &beta, C, N);
}

template<>
void BlitzGPUGemm(const bool transa, const bool transb,
  const int M, const int N, const int K,
  double* A, double* B, double* C, double alpha, double beta) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int ldb = transb ? K : N;
  hipblasDgemm(CuBlasHandle::GetInstance(), TransB, TransA, N, M, K,
    &alpha, B, ldb, A, lda, &beta, C, N);
}

template<>
void BlitzGPUTrans(const int M, const int N, float* input, float* output) {
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasSgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
void BlitzGPUTrans(const int M, const int N, double* input, double* output) {
  const double alpha = 1.0f;
  const double beta = 0.0f;
  hipblasDgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
float BlitzGPUASum(const int N, const float* data) {
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  float sum = 0.0f;
  hipblasSasum(handle, N, data, 1, &sum);
  return sum;
}

template<>
double BlitzGPUASum(const int N, const double* data) {
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  double sum = 0.0f;
  hipblasDasum(handle, N, data, 1, &sum);
  return sum;
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, float* data,
  const int size, const float loc, const float scale) {
  hiprandGenerateNormal(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, double* data,
  const int size, const double loc, const double scale) {
  hiprandGenerateNormalDouble(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen,
  float* data, const int size) {
  hiprandGenerateUniform(*gen, data, size);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen,
  double* data, const int size) {
  hiprandGenerateUniformDouble(*gen, data, size);
}

}  // namespace blitz

