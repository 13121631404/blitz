#include "utils/blitz_gpu_function.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>

namespace blitz {

boost::scoped_ptr<hipblasHandle_t> CuBlasHandle::instance_(0);
boost::once_flag CuBlasHandle::flag_ = BOOST_ONCE_INIT;

template<>
void BlitzGPUGemm(
  float* A, float* B, float* C,
  bool transa, bool transb,
  float alpha, float beta,
  size_t M, size_t N, size_t K) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  size_t lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  size_t ldb = transb ? K : N;
  hipblasSgemm(CuBlasHandle::GetInstance(),
    TransB, TransA,
    N, M, K,
    &alpha,
    B, ldb,
    A, lda,
    &beta,
    C, N);
}

template<>
void BlitzGPUGemm(
  double* A, double* B, double* C,
  bool transa, bool transb,
  double alpha, double beta,
  size_t M, size_t N, size_t K) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  size_t lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  size_t ldb = transb ? K : N;
  hipblasDgemm(CuBlasHandle::GetInstance(),
    TransB, TransA,
    N, M, K,
    &alpha,
    B, ldb,
    A, lda,
    &beta,
    C, N);
}

template<>
void BlitzGPUTrans(float* input, float* output, size_t M, size_t N) {
  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
void BlitzGPUTrans(double* input, double* output, size_t M, size_t N) {
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
float BlitzGPUASum(const float* data, size_t N) {
  float sum = 0.0;
  hipblasSasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
double BlitzGPUASum(const double* data, size_t N) {
  double sum = 0.0;
  hipblasDasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, float* data,
  float loc, float scale, size_t size) {
  hiprandGenerateNormal(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, double* data,
  double loc, double scale, size_t size) {
  hiprandGenerateNormalDouble(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen, float* data, size_t size) {
  hiprandGenerateUniform(*gen, data, size);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen, double* data, size_t size) {
  hiprandGenerateUniformDouble(*gen, data, size);
}

}  // namespace blitz

