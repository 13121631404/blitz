#include "utils/blitz_gpu_function.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>

namespace blitz {

boost::scoped_ptr<hipblasHandle_t> CuBlasHandle::instance_(0);
boost::once_flag CuBlasHandle::flag_ = BOOST_ONCE_INIT;

template<>
void BlitzGPUGemm(
  bool transa, bool transb,
  int M, int N, int K,
  float* A, float* B, float* C,
  float alpha, float beta) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int ldb = transb ? K : N;
  hipblasSgemm(CuBlasHandle::GetInstance(), TransB, TransA, N, M, K,
    &alpha, B, ldb, A, lda, &beta, C, N);
}

template<>
void BlitzGPUGemm(
  bool transa, bool transb,
  int M, int N, int K,
  double* A, double* B, double* C,
  double alpha, double beta) {
  hipblasOperation_t TransA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int lda = transa ? M : K;
  hipblasOperation_t TransB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int ldb = transb ? K : N;
  hipblasDgemm(CuBlasHandle::GetInstance(), TransB, TransA, N, M, K,
    &alpha, B, ldb, A, lda, &beta, C, N);
}

template<>
void BlitzGPUTrans(int M, int N, float* input, float* output) {
  const float alpha = 1.0;
  const float beta = 0.0;
  hipblasSgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
void BlitzGPUTrans(int M, int N, double* input, double* output) {
  const double alpha = 1.0;
  const double beta = 0.0;
  hipblasDgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
float BlitzGPUASum(int N, const float* data) {
  float sum = 0.0;
  hipblasSasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
double BlitzGPUASum(int N, const double* data) {
  double sum = 0.0;
  hipblasDasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, float* data,
  int size, float loc, float scale) {
  hiprandGenerateNormal(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, double* data,
  int size, double loc, double scale) {
  hiprandGenerateNormalDouble(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen,
  float* data, int size) {
  hiprandGenerateUniform(*gen, data, size);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen,
  double* data, int size) {
  hiprandGenerateUniformDouble(*gen, data, size);
}

}  // namespace blitz

