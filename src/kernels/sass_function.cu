#include "kernels/sass_function.h"

#include <hip/hip_runtime.h>

#include <string>

#include "utils/blitz_math_function.h"

namespace blitz {

scoped_ptr<CubinLoadModule> CubinModule::instance_(0);
boost::once_flag CubinModule::flag_ = BOOST_ONCE_INIT;

template<>
void BlitzSassGemm(
  bool transa, bool transb,
  int M, int N, int K,
  const float* A,
  const float* B,
  float* C,
  float alpha,
  float beta) {
  hipFunction_t function;
  int lda, ldb, ldc = N;

#ifdef BLITZ_PERFORMANCE
  float elapsed_time = 0.0f;
  hipEvent_t event_start, event_stop;
  hipEventCreateWithFlags(&event_start, hipEventBlockingSync);
  hipEventCreateWithFlags(&event_stop, hipEventBlockingSync);
  hipEventRecord(event_start, NULL);
#endif  // BLITZ_PERFORMANCE
  // create kernel
  string kernel;
  if (transa == true && transb == false) {
    lda = M * 32;
    ldb = N * 32;
    if (M % 4 == 0 && N % 4 == 0) {
      kernel = "sgemm_tn_128x128_vec";
    } else {
      kernel = "sgemm_tn_128x128";
    }
  } else if (transa == false && transb == true) {
    lda = K;
    ldb = K;
    if (K % 4 == 0) {
      kernel = "sgemm_nt_128x128_vec";
    } else {
      kernel = "sgemm_nt_128x128";
    }
  } else if (transa == false && transb == false) {
    lda = K;
    ldb = N * 32;
    if (K % 4 == 0 && N % 4 == 0) {
      kernel = "sgemm_nn_128x128_vec";
    } else {
      kernel = "sgemm_nn_128x128";
    }
  } else {
    LOG(FATAL) << "Not support both matrice transport!";
  }

  // kernel call, asynrhonize
  function = CubinModule::GetFunction(kernel);

#ifdef BLITZ_PERFORMANCE
  hipEventRecord(event_stop, NULL);
  hipEventSynchronize(event_stop);
  hipEventElapsedTime(&elapsed_time, event_start, event_stop);
  LOG(INFO) << "Load kernel: " << kernel;
  LOG(INFO) << "Load kernel time: " << elapsed_time / 1000.0;
#endif  // BLITZ_PERFORMANCE

  void* params[] = {&A, &B, &C, &alpha, &beta, &lda, &ldb, &ldc,
    (void*)&M, (void*)&N, (void*)&K};
  // TODO(keren): multiple kernels
  int sizeA = 128, sizeB = 128;
  int gridA = M / sizeA + (M % sizeA != 0);
  int gridB = N / sizeB + (N % sizeB != 0);
  // TODO(keren): adjust number of threads
  int threads = 256;

  // lanuch kernel
  hipModuleLaunchKernel(function, 1, gridA, gridB, threads, 1, 1, 0, 0, params, NULL);
}

template<>
void BlitzSassGemm(
  bool transa, bool transb,
  int M, int N, int K,
  const double* A,
  const double* B,
  double* C,
  double alpha,
  double beta) {
  LOG(FATAL) << "sass kernel dost not support double precision";
}

template<>
void BlitzSassConvolution2D(
  const string& phase, 
  int N,
  int C, int H, int W,
  int R, int S,
  int K, int P, int Q,
  int str_h, int str_w,
  int pad_h, int pad_w,
  float* I,
  float* O,
  float* F) {
  float alpha = 1.0f;
  unsigned int D = 1, M = 1, T = 1;
  unsigned int str_d = 1;
  unsigned int pad_d = 0;
  unsigned int WN, HW, DHW, HWN, DHWN;
  unsigned int RS, RST, KRST, CRST;
  unsigned int PQ, QN, MPQ, PQN, MPQN;
  unsigned int magic_HW, shift_HW;
  unsigned int magic_W, shift_W;
  unsigned int magic_RST, shift_RST;
  unsigned int magic_RS, shift_RS;
  unsigned int magic_S, shift_S;
  unsigned int magic_PQ, shift_PQ;
  unsigned int magic_Q, shift_Q;
  unsigned int magic_PQu, shift_PQu;
  unsigned int magic_Qu, shift_Qu;
  unsigned int magic_str_w, shift_str_w;
  unsigned int magic_str_h, shift_str_h;
  unsigned int magic_str_d, shift_str_d;
  unsigned int grid_P = 1;
  unsigned int grid_Q = 1;
  unsigned int grid_PQ = grid_P * grid_Q;
  unsigned int grid_PQM = grid_PQ * M;
  unsigned int CRST32, MPQN32;
  // input
  WN = W * N;
  HW = H * W;
  DHW = D * HW;
  HWN = H * WN;
  DHWN = HWN;
  // filter
  RS = R * S;
  RST = RS;
  KRST = K * RST;
  CRST = C * RST;
  // output
  QN = Q * N;
  PQ = P * Q;
  PQN = P * QN;
  MPQ = PQ;
  MPQN = PQN;
  // magic numbers
  blitz_magic32(DHW, HW, magic_HW, shift_HW);
  blitz_magic32(HW, W, magic_W, shift_W);
  blitz_magic32(CRST, RST, magic_RST, shift_RST);
  blitz_magic32(RST + 32, RS, magic_RS, shift_RS);
  blitz_magic32(RS + 32, S, magic_S, shift_S);
  blitz_magic32(MPQ, PQ, magic_PQ, shift_PQ);
  blitz_magic32(PQ, Q, magic_Q, shift_Q);
  blitz_magic32(grid_PQM, grid_PQ, magic_PQu, shift_PQu);
  blitz_magic32(grid_PQ, grid_Q, magic_Qu, shift_Qu);
  blitz_magic32(W + S - pad_w - 2, str_w, magic_str_w, shift_str_w);
  blitz_magic32(H + R - pad_h - 2, str_h, magic_str_h, shift_str_h);
  blitz_magic32(D + T - pad_d - 2, str_d, magic_str_d, shift_str_d);
  // test param set up
  float *test_param;
#ifdef BLITZ_DEVELOP
  hipError_t cuda_error;
  cuda_error = hipMalloc((void**)&test_param, sizeof(float) * 1024);
  hipMemset(test_param, 0, sizeof(float) * 1024);
#endif
  // arguments
  unsigned int gridX, gridY, gridZ;
  hipError_t result;
  hipFunction_t function;
  string kernel_name;
  if (phase == "forward") {
    void *args[37] = {
      &test_param, &O, &I, &F, &alpha,
      &N, &K, &D, &H, &W, &WN, &HWN, &DHWN,
      &C, &KRST, &RST,
      &RS, &magic_RS, &shift_RS,
      &S, &magic_S, &shift_S,
      &pad_d, &pad_h, &pad_w,
      &str_d, &str_h, &str_w,
      &Q, &PQ, &QN, &PQN, &MPQN,
      &magic_Q, &shift_Q,
      &magic_PQ, &shift_PQ};
    std::cout << "N " << N << std::endl;
    std::cout << "K " << K << std::endl;
    std::cout << "D " << D << std::endl;
    std::cout << "H " << H << std::endl;
    std::cout << "W " << W << std::endl;
    std::cout << "WN " << WN << std::endl;
    std::cout << "HWN " << HWN << std::endl;
    std::cout << "C " << C << std::endl;
    std::cout << "KRST " << KRST << std::endl;
    std::cout << "RST " << RST << std::endl;
    std::cout << "magic_RS " << magic_RS << "shift_RS " << shift_RS << std::endl;
    std::cout << "magic_S " << magic_S << "shift_S " << shift_S << std::endl;
    std::cout << "pad_d " << pad_d << std::endl;
    std::cout << "pad_w " << pad_w << std::endl;
    std::cout << "pad_h " << pad_h << std::endl;
    std::cout << "str_d " << str_d << std::endl;
    std::cout << "str_w " << str_w << std::endl;
    std::cout << "str_h " << str_h << std::endl;
    std::cout << "Q " << Q << std::endl;
    std::cout << "PQ " << PQ << std::endl;
    std::cout << "QN " << QN << std::endl;
    std::cout << "PQN " << PQN << std::endl;
    std::cout << "MPQN " << MPQN << std::endl;
    std::cout << "magic_Q " << magic_Q << "shift_Q " << shift_Q << std::endl;
    std::cout << "magic_PQ " << magic_PQ << "shift_PQ " << shift_PQ << std::endl;
    gridX = MPQ;
    gridY = K / 64 + (K % 64 != 0);
    gridZ = N / 64 + (N % 64 != 0);
    kernel_name = "sconv_fprop_K64_N64";
    // TODO(keren): tune kernels in future
    function = CubinModule::GetFunction(kernel_name);
    result = hipModuleLaunchKernel(function, gridX, gridY, gridZ,
      64, 1, 1, 64 * 8 * 4 + RST * 4 * 2 + 8, 0, args, NULL);
    if (result != hipSuccess) {
      LOG(FATAL) << "Launch kernel: " << kernel_name << " error!";
    }
  } else if (phase == "backward") {
    if (C % 64 == 0) {  // C64
      void *args[45] = {
        &test_param, &I, &O, &F, &alpha,
        &N, &C, &M, &P, &Q, &QN, &PQN, &MPQN,
        &K, &CRST, &RST,
        &RS, &magic_RS, &shift_RS,
        &S, &magic_S, &shift_S,
        &pad_d, &pad_h, &pad_w,
        &str_d, &str_h, &str_w,
        &W, &HW, &WN, &HWN, &DHWN,
        &magic_W, &shift_W,
        &magic_HW, &shift_HW,
        &R, &T,
        &magic_str_w, &shift_str_w,
        &magic_str_h, &shift_str_h,
        &magic_str_d, &shift_str_d};
      gridX = DHW;
      gridY = C / 64 + (C % 64 != 0);
      gridZ = N / 64 + (N % 64 != 0);
      kernel_name = "sconv_bprop_C64_N64";
      function = CubinModule::GetFunction(kernel_name);
      result = hipModuleLaunchKernel(function, gridX, gridY, gridZ,
        64, 1, 1, 0, 0, args, NULL);
      if (result != hipSuccess) {
        LOG(FATAL) << "Launch kernel: " << kernel_name << " error!";
      }
    } else {  // C1
      void *args[41] = {
        &test_param, &I, &O, &F, &alpha,
        &N, &K, &D, &H, &W, &WN, &HWN, &DHWN,
        &C, &CRST,
        &RST, &magic_RST, &shift_RST,
        &RS, &magic_RS, &shift_RS,
        &S, &magic_S, &shift_S,
        &pad_d, &pad_h, &pad_w,
        &str_d, &str_h, &str_w,
        &Q, &PQ, &QN, &PQN, &MPQN,
        &magic_Q, &shift_Q,
        &magic_PQ, &shift_PQ,
        &CRST32,
        &MPQN32};
      gridX = MPQ;
      gridY = CRST / 32 + (CRST % 32 != 0);
      gridZ = N / 64 + (N % 64 != 0);
      kernel_name = "sconv_bprop_C1_N64";
      function = CubinModule::GetFunction(kernel_name);
      result = hipModuleLaunchKernel(function, gridX, gridY, gridZ,
        32, 1, 1, 0, 0, args, NULL);
      if (result != hipSuccess) {
        LOG(FATAL) << "Launch kernel: " << kernel_name << " error!";
      }
    }
  } else if (phase == "update") {
    void *args[43] = {
      &test_param, &F, &I, &O, &alpha,
      &N, &K, &D, &H, &W, &WN, &HWN, &DHWN,
      &C, &CRST,
      &RST, &magic_RST, &shift_RST,
      &RS, &magic_RS, &shift_RS,
      &S, &magic_S, &shift_S,
      &pad_d, &pad_h, &pad_w,
      &str_d, &str_h, &str_w,
      &P, &Q, &PQ, &QN, &PQN, &MPQN,
      &magic_Qu, &shift_Qu,
      &magic_PQu, &shift_PQu,
      &grid_P, &grid_Q, &grid_PQ};
    gridX = grid_PQM;
    gridY = CRST / 128 + (CRST % 128 != 0);
    gridZ = K / 128 + (K % 128 != 0);
    kernel_name = "sconv_update_C128_K128";
    function = CubinModule::GetFunction(kernel_name);
    result = hipModuleLaunchKernel(function, gridX, gridY, gridZ,
      256, 1, 1, 0, 0, args, NULL);
    if (result != hipSuccess) {
      LOG(FATAL) << "Launch kernel: " << kernel_name << " error!";
    }
  }
}

template<>
void BlitzSassConvolution2D(
  const string& phase, 
  int N,
  int C, int H, int W,
  int R, int S,
  int K, int P, int Q,
  int str_h, int str_w,
  int pad_h, int pad_w,
  double* I,
  double* O,
  double* F) {
  LOG(FATAL) << "sass kernel dost not support double precision";
}

}  // namespace blitz

