#include "backend/gpu_backend.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipDNN.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <string>
#include <vector>

#include "util/common.h"
#include "util/blitz_gpu_function.h"
#include "kernels/sass_function.h"

namespace blitz {

#include "backend/gpu_backend_common-inl.h"
#include "backend/gpu_backend_conv-inl.h"
#include "backend/gpu_backend_pack-inl.h"
#include "backend/gpu_backend_pool-inl.h"

INSTANTIATE_BACKEND(GPUTensor);

}  // namespace blitz
