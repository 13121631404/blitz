#include "utils/blitz_gpu_function.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

namespace blitz {

boost::scoped_ptr<hipblasHandle_t> CuBlasHandle::instance_(0);
boost::once_flag CuBlasHandle::flag_ = BOOST_ONCE_INIT;

template<>
void BlitzGPUTrans(float* input, float* output, size_t M, size_t N) {
  float alpha = 1.0;
  float beta = 0.0;
  hipblasSgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
void BlitzGPUTrans(double* input, double* output, size_t M, size_t N) {
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgeam(CuBlasHandle::GetInstance(), HIPBLAS_OP_T, HIPBLAS_OP_N,
    M, N, &alpha, input, N, &beta, input, M, output, M);
}

template<>
float BlitzGPUASum(const float* data, size_t N) {
  float sum = 0.0;
  hipblasSasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
double BlitzGPUASum(const double* data, size_t N) {
  double sum = 0.0;
  hipblasDasum(CuBlasHandle::GetInstance(), N, data, 1, &sum);
  return sum;
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, float* data,
  float loc, float scale, size_t size) {
  hiprandGenerateNormal(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateNormal(hiprandGenerator_t* gen, double* data,
  double loc, double scale, size_t size) {
  hiprandGenerateNormalDouble(*gen, data, size, loc, scale);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen, float* data, size_t size) {
  hiprandGenerateUniform(*gen, data, size);
}

template<>
void BlitzGenerateUniform(hiprandGenerator_t* gen, double* data, size_t size) {
  hiprandGenerateUniformDouble(*gen, data, size);
}

}  // namespace blitz

